#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024  
#define WORD_LEN 32 

__global__ void countWordOccurrences(char *sentence, char *word, int *count, int sentenceLength, int wordLength) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (i <= sentenceLength - wordLength) {
        int match = 1;
        for (int j = 0; j < wordLength; j++) {
            if (sentence[i + j] != word[j]) {
                match = 0;
                break;
            }
        }
        if (match) {
            atomicAdd(count, 1);
        }
    }
}

int main() {
    char sentence[] = "hello world hello hello CUDA hello";
    char word[] = "hello";
    int h_count = 0, *d_count;
    
    int sentenceLength = strlen(sentence);
    int wordLength = strlen(word);

    char *d_sentence, *d_word;
    
    hipMalloc((void**)&d_sentence, sentenceLength * sizeof(char));
    hipMalloc((void**)&d_word, wordLength * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));

    hipMemcpy(d_sentence, sentence, sentenceLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, wordLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &h_count, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (sentenceLength + blockSize - 1) / blockSize;
    
    countWordOccurrences<<<numBlocks, blockSize>>>(d_sentence, d_word, d_count, sentenceLength, wordLength);
    
    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word '%s' appears %d times in the given sentence.\n", word, h_count);

    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_count);

    return 0;
}
