#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024  

__global__ void generateT(char *d_Sin, char *d_T, int Sin_length) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (i < Sin_length) {
        int repeat = i + 1;
        for (int j = 0; j < repeat; j++) {
            d_T[i * repeat + j] = d_Sin[i];
        }
    }
}

int main() {
    char h_Sin[] = "Hai";
    char h_T[N];
    char *d_Sin, *d_T;

    int Sin_length = strlen(h_Sin);
    int T_length = (Sin_length * (Sin_length + 1)) / 2;

    hipMalloc((void**)&d_Sin, Sin_length * sizeof(char));
    hipMalloc((void**)&d_T, T_length * sizeof(char));

    hipMemcpy(d_Sin, h_Sin, Sin_length * sizeof(char), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (Sin_length + blockSize - 1) / blockSize;
    
    generateT<<<numBlocks, blockSize>>>(d_Sin, d_T, Sin_length);
    
    hipMemcpy(h_T, d_T, T_length * sizeof(char), hipMemcpyDeviceToHost);

    h_T[T_length] = '\0';

    printf("Output string T: %s\n", h_T);

    hipFree(d_Sin);
    hipFree(d_T);

    return 0;
}
