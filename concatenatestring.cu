#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024  

__global__ void repeatString(char *d_S, char *d_Sout, int S_length, int repeatCount) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (i < S_length * repeatCount) {
        d_Sout[i] = d_S[i % S_length];
    }
}

int main() {
    char h_S[] = "Hello";
    int repeatCount = 3;
    char h_Sout[N];
    char *d_S, *d_Sout;

    int S_length = strlen(h_S);
    int Sout_length = S_length * repeatCount;

    hipMalloc((void**)&d_S, S_length * sizeof(char));
    hipMalloc((void**)&d_Sout, Sout_length * sizeof(char));

    hipMemcpy(d_S, h_S, S_length * sizeof(char), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (Sout_length + blockSize - 1) / blockSize;
    
    repeatString<<<numBlocks, blockSize>>>(d_S, d_Sout, S_length, repeatCount);
    
    hipMemcpy(h_Sout, d_Sout, Sout_length * sizeof(char), hipMemcpyDeviceToHost);

    h_Sout[Sout_length] = '\0';

    printf("Output string Sout: %s\n", h_Sout);

    hipFree(d_S);
    hipFree(d_Sout);

    return 0;
}
