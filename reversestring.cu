#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define N 1024  

__global__ void reverseWords(char *sentence, char *result, int len) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (i < len) {
        int start = i, end = i;
        
        while (start > 0 && sentence[start - 1] != ' ') start--;
        while (end < len && sentence[end] != ' ') end++;
        
        result[i] = sentence[start + (end - start - 1) - (i - start)];
    }
}

int main() {
    char h_sentence[] = "CUDA programming is fun";
    char h_result[N];
    char *d_sentence, *d_result;
    int len = strlen(h_sentence);

    hipMalloc((void**)&d_sentence, len * sizeof(char));
    hipMalloc((void**)&d_result, len * sizeof(char));

    hipMemcpy(d_sentence, h_sentence, len * sizeof(char), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (len + blockSize - 1) / blockSize;
    
    reverseWords<<<numBlocks, blockSize>>>(d_sentence, d_result, len);
    
    hipMemcpy(h_result, d_result, len * sizeof(char), hipMemcpyDeviceToHost);

    printf("Reversed Words: %s\n", h_result);

    hipFree(d_sentence);
    hipFree(d_result);

    return 0;
}
