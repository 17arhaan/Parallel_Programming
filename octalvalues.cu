#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int to_octal(int num) {
    int octal = 0, place = 1;
    while (num > 0) {
        octal += (num % 8) * place;
        num /= 8;
        place *= 10;
    }
    return octal;
}

__global__ void convert_to_octal(int *input, int *output, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        output[i] = to_octal(input[i]);
    }
}

int main() {
    int N;
    printf("Enter the number of elements: ");
    scanf("%d", &N);

    int h_input[N], h_output[N];

    printf("Enter %d decimal numbers:\n", N);
    for (int i = 0; i < N; i++) {
        scanf("%d", &h_input[i]);
    }

    int *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, N * sizeof(int));

    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    convert_to_octal<<<gridSize, blockSize>>>(d_input, d_output, N);

    hipMemcpy(h_output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Octal Values:\n");
    for (int i = 0; i < N; i++) {
        printf("%d -> %d\n", h_input[i], h_output[i]);
    }

    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
