#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#define THREADS_PER_BLOCK 256
__global__ void computeSine(float *input, float *output, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        output[i] = sinf(input[i]);
    }
}
int main() {
    int N;
    printf("Enter Size : ");
    scanf("%d", &N);
    float *input, *output;
    float *d_input, *d_output;
    size_t size = N * sizeof(float);
    input = (float*)malloc(size);
    output = (float*)malloc(size);
    printf("Enter %d Elements (In rad): ", N);
    for (int i = 0; i < N; i++) {
        scanf("%f", &input[i]);
    }
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);
    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    computeSine<<<blocks, THREADS_PER_BLOCK>>>(d_input, d_output, N);
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);
    printf("Sine values of the input angles:\n[ ");
    for (int i = 0; i < (N < 10 ? N : 10); i++) {
        printf("%f ", output[i]);
    }
    printf("]\n");
    free(input);
    free(output);
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
