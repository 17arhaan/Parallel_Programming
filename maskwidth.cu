#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16  

__global__ void convolution_1D(float *N, float *M, float *P, int Mask_Width, int Width) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float Pvalue = 0;
    int N_start_point = i - (Mask_Width / 2);

    for (int j = 0; j < Mask_Width; j++) {
        if (N_start_point + j >= 0 && N_start_point + j < Width) {
            Pvalue += N[N_start_point + j] * M[j];
        }
    }
    P[i] = Pvalue;
}

int main() {
    int Width, Mask_Width;

    // Get input from the user
    printf("Enter Width: ");
    scanf("%d", &Width);
    
    printf("Enter Mask_Width: ");
    scanf("%d", &Mask_Width);

    float h_N[Width], h_M[Mask_Width], h_P[Width];

    printf("Enter %d Elements:\n", Width);
    for (int i = 0; i < Width; i++) {
        scanf("%f", &h_N[i]);
    }

    printf("Enter %d Elements for the mask array:\n", Mask_Width);
    for (int i = 0; i < Mask_Width; i++) {
        scanf("%f", &h_M[i]);
    }

    float *d_N, *d_M, *d_P;

    hipMalloc(&d_N, Width * sizeof(float));
    hipMalloc(&d_M, Mask_Width * sizeof(float));
    hipMalloc(&d_P, Width * sizeof(float));

    hipMemcpy(d_N, h_N, Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, Mask_Width * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = BLOCK_SIZE;
    int gridSize = (Width + blockSize - 1) / blockSize;

    convolution_1D<<<gridSize, blockSize>>>(d_N, d_M, d_P, Mask_Width, Width);

    hipMemcpy(h_P, d_P, Width * sizeof(float), hipMemcpyDeviceToHost);

    printf("Resultant Array---> ");
    for (int i = 0; i < Width; i++) {
        printf("%f ", h_P[i]);
    }
    printf("\n");

    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);

    return 0;
}
