#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024  

__global__ void generateRS(char *d_S, char *d_RS, int S_length) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < S_length) {
        d_RS[2 * i] = d_S[i];
        d_RS[2 * i + 1] = d_S[i];
    }
}

int main() {
    char h_S[] = "PCAP";
    char h_RS[2 * N];
    char *d_S, *d_RS;

    int S_length = strlen(h_S);
    int RS_length = 2 * S_length;

    hipMalloc((void**)&d_S, S_length * sizeof(char));
    hipMalloc((void**)&d_RS, RS_length * sizeof(char));

    hipMemcpy(d_S, h_S, S_length * sizeof(char), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (S_length + blockSize - 1) / blockSize;
    
    generateRS<<<numBlocks, blockSize>>>(d_S, d_RS, S_length);
    
    hipMemcpy(h_RS, d_RS, RS_length * sizeof(char), hipMemcpyDeviceToHost);

    h_RS[RS_length] = '\0';

    printf("Output string RS: %s\n", h_RS);

    hipFree(d_S);
    hipFree(d_RS);

    return 0;
}
