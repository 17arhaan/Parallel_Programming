#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#define THREADS_PER_BLOCK 256
__global__ void vectorAdd(float *A, float *B, float *C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}
int main() {
    int N;
    printf("Enter Size : ");
    scanf("%d", &N);
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);
    printf("Input Vector A:\n");
    for (int i = 0; i < N; i++) {
        scanf("%f", &A[i]);
    }
    printf("Input Vector B:\n");
    for (int i = 0; i < N; i++) {
        scanf("%f", &B[i]);
    }
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    vectorAdd<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_A, d_B, d_C, N);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printf("Resultant Vector --->\n[ ");
    for (int i = 0; i < (N < 10 ? N : 10); i++) {
        printf("%f ", C[i]);
    }
    printf(" ]\n");
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
