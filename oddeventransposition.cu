#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 8  

__global__ void odd_even_sort(int *arr, int n) {
    int tid = threadIdx.x;

    for (int i = 0; i < n; i++) {
        if (i % 2 == 0 && tid < n / 2) {
            int idx = 2 * tid;
            if (idx + 1 < n && arr[idx] > arr[idx + 1]) {
                int temp = arr[idx];
                arr[idx] = arr[idx + 1];
                arr[idx + 1] = temp;
            }
        }
        __syncthreads();

        if (i % 2 == 1 && tid < (n - 1) / 2) {
            int idx = 2 * tid + 1;
            if (idx + 1 < n && arr[idx] > arr[idx + 1]) {
                int temp = arr[idx];
                arr[idx] = arr[idx + 1];
                arr[idx + 1] = temp;
            }
        }
        __syncthreads();
    }
}

int main() {
    int N;
    
    printf("Enter the number of elements in the array: ");
    scanf("%d", &N);

    int h_arr[N];
    
    printf("Enter %d elements:\n", N);
    for (int i = 0; i < N; i++) {
        scanf("%d", &h_arr[i]);
    }

    int *d_arr;
    hipMalloc(&d_arr, N * sizeof(int));
    hipMemcpy(d_arr, h_arr, N * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = (N + 1) / 2;  
    odd_even_sort<<<1, blockSize>>>(d_arr, N);

    hipMemcpy(h_arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted Array:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    hipFree(d_arr);
    return 0;
}
