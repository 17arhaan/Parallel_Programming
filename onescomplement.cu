#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int ones_complement(int num) {
    int bits = sizeof(int) * 8; // Find number of bits in an int
    int mask = (1 << bits) - 1; // Create a mask of all 1s
    return (~num) & mask;  // Apply bitwise NOT and mask to avoid sign extension
}

__global__ void compute_ones_complement(int *input, int *output, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        output[i] = ones_complement(input[i]);
    }
}

void print_binary(int num) {
    for (int i = 31; i >= 0; i--) {
        printf("%d", (num >> i) & 1);
    }
}

int main() {
    int N;
    
    printf("Enter the number of binary numbers: ");
    scanf("%d", &N);

    int h_input[N], h_output[N];

    printf("Enter %d binary numbers (as decimal integers, e.g., 10 for 0b1010):\n", N);
    for (int i = 0; i < N; i++) {
        scanf("%d", &h_input[i]);
    }

    int *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, N * sizeof(int));

    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    compute_ones_complement<<<gridSize, blockSize>>>(d_input, d_output, N);

    hipMemcpy(h_output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nOne's Complement Results:\n");
    for (int i = 0; i < N; i++) {
        printf("Original: ");
        print_binary(h_input[i]);
        printf(" (%d) -> One's Complement: ", h_input[i]);
        print_binary(h_output[i]);
        printf(" (%d)\n", h_output[i]);
    }

    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
